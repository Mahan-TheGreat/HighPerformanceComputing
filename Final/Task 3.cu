#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <time.h>

//Compile with: nvcc -o cudaCrack Task3.cu
 
//Password Cracking using CUDA 
__device__ char* encryptDecrypt(char* tempPassword){

	char * generatedPwd = (char *) malloc(sizeof(char) * 11);

	generatedPwd[0] = tempPassword[0] + 2;
	generatedPwd[1] = tempPassword[0] - 2;
	generatedPwd[2] = tempPassword[0] + 1;
	generatedPwd[3] = tempPassword[1] + 3;
	generatedPwd[4] = tempPassword[1] - 3;
	generatedPwd[5] = tempPassword[1] - 1;
	generatedPwd[6] = tempPassword[2] + 2;
	generatedPwd[7] = tempPassword[2] - 2;
	generatedPwd[8] = tempPassword[3] + 4;
	generatedPwd[9] = tempPassword[3] - 4;
	generatedPwd[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //Checking all uppercase letters limits
			if(generatedPwd[i] > 122){
				generatedPwd[i] = (generatedPwd[i] - 122) + 97;
			}else if(generatedPwd[i] < 97){
				generatedPwd[i] = (97 - generatedPwd[i]) + 97;
			}
		}else{ //Checking numbers
			if(generatedPwd[i] > 57){
				generatedPwd[i] = (generatedPwd[i] - 57) + 48;
			}else if(generatedPwd[i] < 48){
				generatedPwd[i] = (48 - generatedPwd[i]) + 48;
			}
		}
	}
	return generatedPwd;
}

//GPU function- can be launced by many blocks and threads
__global__ void crack(char * alphabet, char * numbers){

char matchedPwd[4];

matchedPwd[0] = alphabet[blockIdx.x];
matchedPwd[1] = alphabet[blockIdx.y];

matchedPwd[2] = numbers[threadIdx.x];
matchedPwd[3] = numbers[threadIdx.y];

//Idx --> Index of the block or thread

char* encryptedPwd = "gcfvpr8491"; 
char* search = encryptDecrypt(matchedPwd);

//Matching encrypted password with generated password
int iter = 0;
int is_match = 0;
while (*encryptedPwd != '\0' || *search != '\0') {
	if (*encryptedPwd == *search) {
		encryptedPwd++;
		search++;
	} else if ((*encryptedPwd == '\0' && *search != '\0') || (*encryptedPwd != '\0' && *search == '\0') || *encryptedPwd != *search) {
		is_match = 1;
	
		break;
	}
}
if (is_match == 0) {
	//Print password if encrypted password matches generated password
	printf("Password found successfully: %c%c%c%c \n", matchedPwd[0],matchedPwd[1],matchedPwd[2],matchedPwd[3]);
}


}

int time_count(struct timespec *start, struct timespec *end,
                   long long int *diff)
{
    long long int in_sec = end->tv_sec - start->tv_sec;
    long long int in_nano = end->tv_nsec - start->tv_nsec;
    if (in_nano < 0)
    {
        in_sec--;
        in_nano += 1000000000;
    }
    *diff = in_sec * 1000000000 + in_nano;
    return !(*diff > 0);
}

int main(int argc, char ** argv){

    struct timespec start, end;
    long long int time_used;
     
	//possible characters and digits
	char cpuCharacter[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
	char cpuDigits[26] = {'0','1','2','3','4','5','6','7','8','9'};

	char * gpuCharacter;
	hipMalloc( (void**) &gpuCharacter, sizeof(char) * 26); 
	hipMemcpy(gpuCharacter, cpuCharacter, sizeof(char) * 26, hipMemcpyHostToDevice);

	char * gpuDigits;
	hipMalloc( (void**) &gpuDigits, sizeof(char) * 26); 
	hipMemcpy(gpuDigits, cpuDigits, sizeof(char) * 26, hipMemcpyHostToDevice);
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
	crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuCharacter, gpuDigits );
    hipDeviceSynchronize();

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    time_count(&start, &end, &time_used);

    printf("Time taken: %f seconds OR %lld Nano Seconds\n", (time_used / 1.0e9), time_used);
  
    
	return 0;
}












