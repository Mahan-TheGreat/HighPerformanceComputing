#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

//Compile with:  nvcc -o Blur BoxBlur.cu lodepng.cpp

int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

//GPU Kernel function
__global__ void ImageFilter(unsigned char * cpu_imageOutput, unsigned char * cpu_imageInput,unsigned int width,unsigned int height){
    
	//for rgbt of a color
	int r = 0;
	int g = 0;
	int b = 0;
	int t = 0;
	int x,y;
	int count = 0;

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pixel = idx*4;

	for(x = (pixel - 4); x <=  (pixel + 4); x+=4){
		//pixel check for found at x
		if ((x > 0) && x < (height * width * 4) && ((x-4)/(4*width) == pixel/(4*width))){
			for(y = (x - (4 * width)); y <=  (x + (4 * width)); y+=(4*width)){
				if(y > 0 && y < (height * width * 4)){
					r += cpu_imageInput[y];
					g += cpu_imageInput[1+y];
					b += cpu_imageInput[2+y]; 
					count++;
				}
			}
		}
	}
	
	t = cpu_imageInput[3+pixel];

	cpu_imageOutput[pixel] = r / count;
	cpu_imageOutput[1+pixel] = g / count;
	cpu_imageOutput[2+pixel] = b / count;
	cpu_imageOutput[3+pixel] = t;
}

int main(int argc, char **argv){

	unsigned int errors;
	unsigned int occErrors;
	unsigned char* images;
	unsigned int width;
	unsigned int height;
	struct timespec start, finish;   
  	long long int time_elapsed;
  	clock_gettime(CLOCK_MONOTONIC, &start);
	const char* filename = "Image.png";
	const char* newFileName = "blur Image.png";

	errors = lodepng_decode32_file(&images, &width, &height, filename);
	if(errors){
		printf("Error %u: %s\n", errors, lodepng_error_text(errors));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = images[i];
	}

	// declaring variable
	unsigned char * d_in;
	unsigned char * d_out;

	// allocating GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel function
	ImageFilter<<<height, width>>>(d_out, d_in, width, height);

	//copy result back to cpu
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	
	printf("the blur Image has been created!\n");
	
	occErrors = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(occErrors){
		printf("Error occurred %u: %s\n", errors, lodepng_error_text(occErrors));
	}

	//Free allocated memory 
	hipFree(d_in);
	hipFree(d_out);

	clock_gettime(CLOCK_MONOTONIC, &finish);
  	time_difference(&start, &finish, &time_elapsed);
  	printf("Time elapsed %lldns or %0.9lfs\n", time_elapsed,(time_elapsed/1.0e9)); 
	return 0;
}
